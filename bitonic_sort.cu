
#include <hip/hip_runtime.h>
#include <cstdio>



extern "C" {

__device__
void min_max(int* for_min, int* for_max) {
	int min = *for_min;
	int max = *for_max;
	if (max < min) {
		atomicExch(for_max, min);
		atomicExch(for_min, max);
	}
};


__global__
void bitonic_sort(int* in, int n) {
	int thid = blockIdx.x * blockDim.x + threadIdx.x;	
		
	int d_traingle;
	int local_thid;
	int opposite;


	for (d_traingle = 2; d_traingle <= n; d_traingle*=2) {
		local_thid = thid % d_traingle;		
		opposite = thid - local_thid + d_traingle - 1 - local_thid;
		if (local_thid < d_traingle/2) {
			min_max(in + thid, in + opposite);
		}

		__syncthreads();

		for (int d = d_traingle/2; d >= 2; d /= 2) {
			local_thid = thid % d;	
			if (local_thid < d/2) {
				opposite = thid + d/2;
				min_max(in + thid, in + opposite);
			}
			__syncthreads();
		}
		__syncthreads();
	}

}

__global__
void bitonic_merge(int* in, int d) {
	int thid = blockIdx.x * blockDim.x + threadIdx.x;	
 	int local_thid = thid % d;	
 	int opposite = thid + d/2;
	if (local_thid < d/2) {
		min_max(in + thid, in + opposite);
	}
}

__global__
void bitonic_triangle_merge(int* in, int d_traingle) {
 	int thid = blockIdx.x * blockDim.x + threadIdx.x;
 	// printf("%d %d traingle thid \n", d_traingle, thid);	
	int local_thid = thid % d_traingle;		
	int opposite = thid - local_thid + d_traingle - 1 - local_thid;
	if (local_thid < d_traingle/2) {
		min_max(in + thid, in + opposite);
	}
}



}



